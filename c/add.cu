
#include <hip/hip_runtime.h>
#include <stdio.h>
// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main()
{
  int N = 1000000; // 1M elements

  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0;
    y[i] = 2.0;
  }

  // Run kernel on 1M elements on the CPU
  add<<<1, 1>>>(N, x, y);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0;
  for(int i = 0; i < N; i++){
    maxError = fmax(maxError, fabs(y[i]-3.0));
  }
  printf("Max error: %f\n",maxError);

  // Free memory
  hipFree(x);
  hipFree(y);

}
